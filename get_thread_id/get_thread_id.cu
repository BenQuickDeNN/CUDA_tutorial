/*********************************************************************
 * @file 	check_gpuinfo.cu
 * @brief 	fetch thread ID
 * @author 	Bin Qu
 * @email 	benquickdenn@foxmail.com
 * @date	2019-12-1
 * you can reedit or modify this file
*********************************************************************/

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <>

#include "kernels.h"

/**
 * @brief display thread ID
 */
void dispThreadID();

/**
 * @brief main entry
 * @return exit status
 */
int main(int argc, char** argv)
{
	dispThreadID();
	return 0;
}

void dispThreadID()
{
	/* initialize grid */
	dim3 gridSize(2, 3, 4);

	/* initialize block */
	dim3 blockSize(5, 6, 7);

	/* allocate memory on host */
	const unsigned int memSpace = gridSize.z * gridSize.y * gridSize.x *
		blockSize.z * blockSize.y * blockSize.x;
	unsigned int* threadID;
	threadID = (unsigned int*)std::malloc(memSpace * sizeof(unsigned int));
	
	/* allocate memory on device */
	unsigned int* cuThreadId;
	hipMalloc((void**)&cuThreadId, memSpace * sizeof(unsigned int));

	/* copy data from host to device */
	/* in this application, there is no need to copy data from host to device */
	//hipMemcpy(cuThreadId, threadID, memSpce * sizeof(unsigned int), hipMemcpyHostToDevice)

	/* call kernel */
	computeThreadID<<<gridSize, blockSize>>>(cuThreadId);

	/* copy data from device to host */
	hipMemcpy(threadID, cuThreadId, memSpace * sizeof(unsigned int), hipMemcpyDeviceToHost);

	/* free memory on device */
	hipFree(cuThreadId);

	/* display thread id */
	for (int i = 0; i < memSpace; i++)
		std::printf("%d, ", threadID[i]);
	std::printf("\r\n");

	/* free memory on host */
	std::free(threadID);
}