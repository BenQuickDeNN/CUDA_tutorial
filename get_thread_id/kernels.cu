#include <hip/hip_runtime.h>


/**
 * @brief cuda kernel -- compute id of a thread
 * @param array that stores thread ids
 * @return return is not allowed
 */
__global__ void computeThreadID(unsigned int* threadID);

__global__ void computeThreadID(unsigned int* threadID)
{
    int tid = (blockIdx.z * gridDim.y * gridDim.x +
        blockIdx.y * gridDim.x + blockIdx.x) *
        blockDim.z * blockDim.y * blockDim.x +
        threadIdx.z * blockDim.y * blockDim.x +
        threadIdx.y * blockDim.x + threadIdx.x;
    threadID[tid] = tid;
}