/*********************************************************************
 * @file 	arithmetic.cu
 * @brief 	arithmetic file, including vadd and mmul
 * @author 	Bin Qu
 * @email 	benquickdenn@foxmail.com
 * @date	2020-3-5
 * you can reedit or modify this file
*********************************************************************/

#include "kernels.h"

#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <>

/**
* @brief main function to activate vadd kernel
* @param c vector c
* @param a vector a
* @param b vector b
* @param alpha scalar alpha
* @param beta scalar beta
* @param batSize batch size
*/
template<class T>
void cuda_vadd(T* c, const T* a, const T* b,
    const T& alpha, const T& beta, const int& batSize, const int& steps, 
    const dim3& gridSize, const dim3& blockSize, const int& len);

template<class T>
void cuda_vadd(T* c, const T* a, const T* b,
    const T& alpha, const T& beta, const int& batSize, const int& steps, 
    const dim3& gridSize, const dim3& blockSize, const int& len)
{
    if (c == nullptr || a == nullptr || b == nullptr)
    {
        std::fprintf(stderr, "vadd error: vector empty!\r\n");
        return;
    }

    /* allocate memory on gpu */
    T *cu_c, *cu_a, *cu_b;

    hipMalloc((void**)&cu_c, len * sizeof(T));
    hipMalloc((void**)&cu_a, len * sizeof(T));
    hipMalloc((void**)&cu_b, len * sizeof(T));

    /* copy data from host to gpu */
    hipMemcpy(cu_a, a, len * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(cu_b, b, len * sizeof(T), hipMemcpyHostToDevice);

    /* activate kernel */
    kernel_vadd<<<gridSize, blockSize>>>(cu_c, cu_a, cu_b, alpha, beta, batSize, steps);

    /* copy data from gpu to host */
    hipMemcpy(c, cu_c, len * sizeof(T), hipMemcpyDeviceToHost);
    
    /* free memory on gpu */
    hipFree(cu_b);
    hipFree(cu_a);
    hipFree(cu_c);
}