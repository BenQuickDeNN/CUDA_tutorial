#pragma once

#include "cuda_lib.cu"
#include "configure.hpp"

/**
 * @brief 启动函数
 * @param _C 储存结果的矩阵
 * @param _A 参与运算的矩阵
 * @param _B 参与运算的矩阵
 * @param _height 矩阵C的高度
 * @param _width 矩阵C的宽度
 * @param _offset_h 矩阵C竖直方向的起始索引
 * @param _width_A 矩阵A的宽度和矩阵B的高度
 * @param _width_B 矩阵B的宽度
 * @param _device_id GPU编号号
 */
void cuda_exec_gemm(type *_C, type *_A, type *_B, 
    size_t _height, size_t _width, 
    size_t _offset_h, size_t _width_A, size_t _width_B,
    size_t _device_id)
{
    hipSetDevice(_device_id); // 设置用于计算的GPU

    /* 设置网格grid和block */
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, 0);
    auto num_blocks = devProp.multiProcessorCount * devProp.maxBlocksPerMultiProcessor;
}