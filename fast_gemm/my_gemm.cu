#include <cstdlib>
#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>

using namespace std;

const size_t GIGA = 1 << 30;

typedef float type;
// typedef double type

const size_t WidthA = 1024;
const size_t HeightA = 1024;

const size_t WidthB = 1024;
const size_t HeightB = 1024;

type A[HeightA * WidthA], B[HeightB * WidthB], C[HeightA * WidthB];

__global__ void cuda_gemm(type *_C, type *_A, type *_B, size_t _wC, size_t _wA, size_t _wB, size_t _maxIdx)
{
    size_t tid = blockIdx.x * blockDim.x + threadIdx.x; // 线程号
    size_t stride = gridDim.x * blockDim.x; // 索引更新步长
    for (size_t idx = tid; idx < _maxIdx; idx += stride)
    {
        size_t y = idx / _wC;
        size_t x = idx % _wC;
        type _c = 0;
        size_t y_wA = y * _wA;
        /* 计算 */
        for (size_t k = 0; k < _wA; ++k)
        {
            _c += _A[y_wA + k] * _B[k * _wB + x];
        }
        _C[idx] = _c;
    }
}

// 使用shared memory
__global__ void cuda_gemm2(type *_C, type *_A, type *_B, size_t _wC, size_t _wA, size_t _wB, size_t _maxIdx)
{
    // extern __shared__ type As[], Bs[]; // 动态分配
    // 获取As的长度
}

bool cuda_exec_gemm(void (*kernel)(type*, type*, type*, size_t, size_t, size_t, size_t), const int &_device_id)
{
    /* 选择设备 */
    if (hipSetDevice(_device_id) != 0)
    {
        cerr << "error: GPU " << _device_id << " is not available" << endl;
        return false;
    }
    cout << "select GPU " << _device_id << endl;

    /* 设置网格grid和block */
    hipDeviceProp_t devProp;
    hipGetDeviceProperties(&devProp, _device_id);
    size_t total_workload = (HeightA * WidthB + HeightA * WidthA + HeightB * WidthB) * sizeof(type);
    std::cout << "total workload is " << (double)total_workload / (double)GIGA << " GB" << std::endl;
    if (total_workload > devProp.totalGlobalMem) // 检查工作负载是否超过内存容量
    {
        std::cerr << "error: too large total workload! Only " << (double)devProp.totalGlobalMem  / (double)GIGA << "GB are available" << std::endl;
        return false;
    }
    int num_blocks = devProp.multiProcessorCount * devProp.maxBlocksPerMultiProcessor;
    int num_threads_per_blocks = devProp.maxThreadsPerBlock;
    dim3 gridSize(num_blocks, 1, 1);
    dim3 blockSize(num_threads_per_blocks, 1, 1);

    /* 计算每个block对应的shared memory容量 */
    size_t sharedMemSize = devProp.sharedMemPerBlock / sizeof(type) / 2;
    cout << "the shared memory size per block is " << devProp.sharedMemPerBlock / 1024 << " KB" << endl;

    /* 分配内存 */
    type *cu_C, *cu_A, *cu_B;
    hipMalloc((void**)&cu_C, HeightA * WidthB * sizeof(type));
    hipMalloc((void**)&cu_A, HeightA * WidthA * sizeof(type));
    hipMalloc((void**)&cu_B, HeightB * WidthB * sizeof(type));

    /* GPU热身 */
    kernel<<<gridSize, blockSize, sharedMemSize>>>(cu_C, cu_A, cu_B, WidthB, WidthA, WidthB, HeightA * WidthB);

    /* 传输数据 */
    hipMemcpy(cu_A, A, HeightA * WidthA * sizeof(type), hipMemcpyHostToDevice);
    hipMemcpy(cu_B, B, HeightB * WidthB * sizeof(type), hipMemcpyHostToDevice);

    /* 执行内核 */
    cout << "start kernel..." << endl;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsed;
    hipEventRecord(start, 0);
    kernel<<<gridSize, blockSize, sharedMemSize>>>(cu_C, cu_A, cu_B, WidthB, WidthA, WidthB, HeightA * WidthB);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed, start, stop);
    cout << "kernel elapsed " << elapsed << " ms, ";
    cout << "the speed is " << (float)(HeightA * WidthB * 2 * WidthA) * 1000 / elapsed / (float)GIGA << " GFlops" << endl;
    hipEventDestroy(stop);
    hipEventDestroy(start);

    /* 传回结果 */
    hipMemcpy(C, cu_C, HeightA * WidthB * sizeof(type), hipMemcpyDeviceToHost);

    /* 释放内存 */
    hipFree(cu_B);
    hipFree(cu_A);
    hipFree(cu_C);

    return true;
}

void myfill(type *arr, const size_t &_height, const size_t &_width, const type &_val)
{
#pragma omp parallel for
    for (size_t h = 0; h < _height; ++h)
    {
        const size_t idx1 = h * _width;
        for (size_t w = 0; w < _width; ++w)
        {
            arr[idx1 + w] = _val;
        }
    }
}

void verify(const type *cu_C, const type *A, const type *B)
{
    type *C = new type[HeightA * WidthB];
#pragma omp parallel for
    for (size_t h = 0; h < HeightA; ++h)
    {
        const size_t idx1 = h * WidthB;
        for (size_t w = 0; w < WidthB; ++w)
        {
            const size_t idx2 = idx1 + w;
            C[idx2] = 0.0;
#pragma unroll
            for (size_t k = 0; k < WidthA; ++k)
            {
                C[idx2] += A[idx1 + k] * B[k * WidthB + w];
            }
        }
    }
    for (size_t h = 0; h < HeightA; ++h)
    {
        const size_t idx1 = h * WidthB;
        for (size_t w = 0; w < WidthB; ++w)
        {
            const size_t idx2 = idx1 + w;
            if (abs(C[idx2] - cu_C[idx2]) > 0.1)
            {
                cerr << "computation error occurs where h = " << h << ", w = " << w;
                cerr <<"; and C = " << C[idx2] << " but cu_C = " << cu_C[idx2] << endl;
                return;
            }
        }
    }
    cout << "computation correct" << endl;
}

int main()
{
    myfill(A, HeightA, WidthA, 2.0);
    myfill(B, HeightB, WidthB, 3.0);
    if (cuda_exec_gemm(cuda_gemm, 1))
    {
        verify(C, A, B);
    }
    else
    {
        cerr << "computation fail!" << endl;
    }
    return 0;
}